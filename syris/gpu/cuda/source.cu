#include "hip/hip_runtime.h"
#include "Commons.cuh"
#include "Ray.cuh"

#define SENTINEL -1
#define INVALID -1

#define INT_MAX 2147483647
#define INT_MIN -2147483648

typedef unsigned int morton_t;
typedef int delta_t;

typedef struct {
    unsigned int nb_keys;
    morton_t *keys;
    unsigned int *indices;
    int *entered;
    int *rope;
    int *left;
    float4 *bboxMin;
    float4 *bboxMax;
} Tree;

__device__ bool isLeaf(Tree &tree, unsigned int index) {
    return index < tree.nb_keys;
}

__device__ unsigned int toInternalRepresentation(Tree &tree, unsigned int index) {
    return index + tree.nb_keys;
}

__device__ void growBox(float4 &bbMinInput, float4 &bbMaxInput, float4 *bbMinOutput, float4 *bbMaxOutput) {
    bbMinOutput->x = fminf(bbMinInput.x, bbMinOutput->x);
    bbMinOutput->y = fminf(bbMinInput.y, bbMinOutput->y);
    bbMinOutput->z = fminf(bbMinInput.z, bbMinOutput->z);

    bbMaxOutput->x = fmaxf(bbMaxInput.x, bbMaxOutput->x);
    bbMaxOutput->y = fmaxf(bbMaxInput.y, bbMaxOutput->y);
    bbMaxOutput->z = fmaxf(bbMaxInput.z, bbMaxOutput->z);
}

__device__ int delta(Tree &tree, int index) {

    if (index < 0 || index >= tree.nb_keys - 1) {
        return INT_MAX;
    }
    
    // TODO: augment the function if the codes are the same
    unsigned int a = tree.keys[index];
    unsigned int b = tree.keys[index + 1];
    int x = a ^ b;
    return x + (!x) * (INT_MIN + (index ^ (index + 1))) - 1; // 
}

__device__ void setRope(Tree &tree, unsigned int skip_index, int range_right, delta_t delta_right) {
    int rope;

    if (range_right != tree.nb_keys - 1) {
        int r = range_right + 1;
        rope = delta_right < delta(tree, r) ? r : toInternalRepresentation(tree, r);
    }
    else {
        rope = SENTINEL;
    }
    tree.rope[skip_index] = rope;
}

__device__ void setLeftChild (Tree &tree, unsigned int parent, unsigned int left_child) {
    tree.left[parent] = left_child;
}

__device__ void setBBMin (Tree &tree, unsigned int parent, float4 bbMin) {
    tree.bboxMin[parent] = bbMin;
}

__device__ void setBBMax (Tree &tree, unsigned int parent, float4 bbMax) {
    tree.bboxMax[parent] = bbMax;
}

__device__ float4 getBBMin (Tree &tree, unsigned int index) {
    return tree.bboxMin[index];
}

__device__ float4 getBBMax (Tree &tree, unsigned int index) {
    return tree.bboxMax[index];
}

__device__ int getRope (Tree &tree, unsigned int index) {
    return tree.rope[index];
}

__device__ int getLeftChild (Tree &tree, unsigned int index) {
    return tree.left[index];
}

__device__ void updateParents(Tree &tree, int i) {
    int range_left = i;
    int range_right = i;
    delta_t delta_left = delta(tree, i - 1);
    delta_t delta_right = delta(tree, i);

    float4 bbMinCurrent = getBBMin (tree, i);
    float4 bbMaxCurrent = getBBMax (tree, i);

    setRope(tree, i, range_right, delta_right);

    unsigned const root = toInternalRepresentation(tree, 0);

    do {
        int left_child;
        int right_child;
        if (delta_right < delta_left) {
            const int apetrei_parent = range_right;

            range_right = atomicCAS (&(tree.entered[toInternalRepresentation(tree, apetrei_parent)]), INVALID, range_left);

            if (range_right == INVALID) {
                return;
            }
            delta_right = delta(tree, range_right);

            left_child = i;

            right_child = apetrei_parent + 1;

            // Memory sync
            __threadfence();

            if (right_child != range_right) {
                right_child = toInternalRepresentation(tree, right_child);
            }

            float4 bbMinRight = getBBMin (tree, right_child);
            float4 bbMaxRight = getBBMax (tree, right_child);
            growBox(bbMinRight, bbMaxRight, &bbMinCurrent, &bbMaxCurrent);
        }
        else {
            int const apetrei_parent = range_left - 1;
            range_left = atomicCAS (&(tree.entered[toInternalRepresentation(tree, apetrei_parent)]), INVALID, range_right);

            if (range_left == INVALID){
                return;
            }

            delta_left = delta(tree, range_left - 1);

            left_child = apetrei_parent;
            bool const left_is_leaf = (left_child == range_left);

            // Memory sync
            __threadfence();
            
            if (!left_is_leaf) {
                left_child = toInternalRepresentation(tree, left_child);
            }

            float4 bbMinLeft = getBBMin (tree, left_child);
            float4 bbMaxLeft = getBBMax (tree, left_child);
            growBox(bbMinLeft, bbMaxLeft, &bbMinCurrent, &bbMaxCurrent);
        }

        int karras_parent = delta_right < delta_left ? range_right : range_left;
        karras_parent = toInternalRepresentation(tree, karras_parent);

        setLeftChild(tree, karras_parent, left_child);
        setBBMin(tree, karras_parent, bbMinCurrent);
        setBBMax(tree, karras_parent, bbMaxCurrent);
        setRope(tree, karras_parent, range_right, delta_right);

        i = karras_parent;
    }
    while (i != root);
    
    return;
}


__device__ void updateParentsVoxelgrid(Tree &tree, int i, int j, int k) {
    int range_left = i;
    int range_right = i;
    delta_t delta_left = delta(tree, i - 1);
    delta_t delta_right = delta(tree, i);

    float4 bbMinCurrent = getBBMin (tree, i);
    float4 bbMaxCurrent = getBBMax (tree, i);

    setRope(tree, i, range_right, delta_right);

    unsigned const root = toInternalRepresentation(tree, 0);

    do {
        int left_child;
        int right_child;
        if (delta_right < delta_left) {
            const int apetrei_parent = range_right;

            range_right = atomicCAS (&(tree.entered[toInternalRepresentation(tree, apetrei_parent)]), INVALID, range_left);

            if (range_right == INVALID) {
                return;
            }
            delta_right = delta(tree, range_right);

            left_child = i;

            right_child = apetrei_parent + 1;

            // Memory sync
            __threadfence();

            if (right_child != range_right) {
                right_child = toInternalRepresentation(tree, right_child);
            }

            float4 bbMinRight = getBBMin (tree, right_child);
            float4 bbMaxRight = getBBMax (tree, right_child);
            growBox(bbMinRight, bbMaxRight, &bbMinCurrent, &bbMaxCurrent);
        }
        else {
            int const apetrei_parent = range_left - 1;
            range_left = atomicCAS (&(tree.entered[toInternalRepresentation(tree, apetrei_parent)]), INVALID, range_right);

            if (range_left == INVALID){
                return;
            }

            delta_left = delta(tree, range_left - 1);

            left_child = apetrei_parent;
            bool const left_is_leaf = (left_child == range_left);

            // Memory sync
            __threadfence();
            
            if (!left_is_leaf) {
                left_child = toInternalRepresentation(tree, left_child);
            }

            float4 bbMinLeft = getBBMin (tree, left_child);
            float4 bbMaxLeft = getBBMax (tree, left_child);
            growBox(bbMinLeft, bbMaxLeft, &bbMinCurrent, &bbMaxCurrent);
        }

        int karras_parent = delta_right < delta_left ? range_right : range_left;
        karras_parent = toInternalRepresentation(tree, karras_parent);

        setLeftChild(tree, karras_parent, left_child);
        setBBMin(tree, karras_parent, bbMinCurrent);
        setBBMax(tree, karras_parent, bbMaxCurrent);
        setRope(tree, karras_parent, range_right, delta_right);

        i = karras_parent;
    }
    while (i != root);
    
    return;
}


__device__ void query (Tree &tree, Ray &ray, CandidateList &candidates) {
    int current_node = toInternalRepresentation(tree, 0);
    
    do {
        float4 bbMax = getBBMax (tree, current_node);
        float4 bbMin = getBBMin (tree, current_node);
        if (ray.intersects(bbMin, bbMax)) {
            if (isLeaf(tree, current_node)) {
                if (candidates.count == MAX_COLLISIONS) {
                    return;
                }
                candidates.collisions[candidates.count++] = current_node;
                current_node = getRope(tree, current_node);
            }
            else {
                current_node = getLeftChild(tree, current_node);
            }
        }
        else {
            current_node = getRope(tree, current_node);
        }
    }
    while (current_node != SENTINEL);
}

__device__ float4 phi (int i, int j, float2 D, uint2 N) {
    float delta_x = D.x / (N.x-1);
    float delta_y = D.y / (N.y-1);
    float Dx2 = D.x / 2;
    float Dy2 = D.y / 2;

    float x = -Dx2 + i * delta_x;
    float y = -Dy2 + j * delta_y;
    return make_float4(x, y, 0.0, 0);
}

__device__ float sumTvalues (CollisionList &t_values) {
    float thickness = 0;
    for (int i = 0; i < t_values.count; i++) {
        thickness += t_values.collisions[i];
    }
    return thickness;
}
//
__device__ float computeThickness(CollisionList &tvalues) {
    float result = 0.0;
    float epsilon = 1e-6;
    int i = 0, j = 1;
    if (tvalues.count == 0) {
        return 0.0;
    }
    
    float t1 = tvalues.collisions[i];
    while (j < tvalues.count) {
        float t2 = tvalues.collisions[j];
        float d = fabsf (t2 - t1);
        if (d > epsilon){
            result += d;
            t1 = t2;
            j++;
        }
        j++;
    }

    return result;
}

inline __device__ float dot(const float4& a, const float4& b) {
    return a.x * b.x + a.y * b.y + a.z * b.z;
}

// printf ("ID -> %d \nT -> \n%f %f %f \n%f %f %f \n%f %f %f\n", primIndex,
//             vertices[primIndex*3].x, vertices[primIndex*3].y, vertices[primIndex*3].z,
//             vertices[primIndex*3+1].x, vertices[primIndex*3+1].y, vertices[primIndex*3+1].z,
//             vertices[primIndex*3+2].x, vertices[primIndex*3+2].y, vertices[primIndex*3+2].z);

__device__ float matchOuterPairs (
        CandidateList &candidates, CollisionList &tvalues, Ray &ray,
        float4* __restrict__ vertices,
        float4* __restrict__ normals,
        Tree &tree) {
    float thickness = 0.0, inner = 0.0;
    int counter = 0;

    if (tvalues.count == 1) {
        return tvalues.collisions[0];
    }

    for (int i = 0; i < tvalues.count; i++) {
        unsigned primIndex = candidates.collisions[i];
        bool is_neg = dot(ray.getDirection(), normals[primIndex]) < 0;
        if (!is_neg && counter == 0) {
            continue;
        }

        if (is_neg) {
            if (counter++ == 0)
                inner = tvalues.collisions[i];
        }
        else {
            if (--counter == 0) {
                thickness += tvalues.collisions[i] - inner;
            }
        }

    }
    return thickness;
}

__device__ float getNbCandidate (Ray &ray, Tree &tree, float4 *vertices, float4 *normals) {
    CandidateList candidates;
    candidates.count = 0;
    memset(candidates.collisions, 0, MAX_COLLISIONS * sizeof(int));

    // This is where the acceleration structure (BVH) is actually usefull
    query(tree, ray, candidates);
    
    return candidates.count;
}

__device__ float getSumedTvalues (Ray &ray, Tree &tree, float4 *vertices) {
    CandidateList candidates;
    candidates.count = 0;
    memset(candidates.collisions, 0, MAX_COLLISIONS * sizeof(int));

    CollisionList tvalues;
    tvalues.count = 0;
    memset(tvalues.collisions, 0, MAX_COLLISIONS * sizeof(float));

    // This is where the acceleration structure (BVH) is actually usefull
    query(tree, ray, candidates);

    if (candidates.count == 0) {
        return 0.0;
    }

    // Test the candidates for actual intersections
    for (int i = 0; i < candidates.count; i++) {
        int primIndex = candidates.collisions[i]*3;
        
        float4 V1 = vertices[primIndex];
        float4 V2 = vertices[primIndex + 1];
        float4 V3 = vertices[primIndex + 2];

        float t;
        if (ray.intersects(V1, V2, V3, t)) {
            candidates.collisions[tvalues.count] = candidates.collisions[i];
            tvalues.collisions[tvalues.count++] = t;
        }
    }

    return sumTvalues(tvalues);
}


__device__ float traceParallelRay (
        Ray &ray, Tree &tree, 
        float4* __restrict__ vertices,
        float4* __restrict__ normals) {
    CandidateList candidates;
    candidates.count = 0;
    memset(candidates.collisions, 0, MAX_COLLISIONS * sizeof(int));

    CollisionList tvalues;
    tvalues.count = 0;
    memset(tvalues.collisions, 0, MAX_COLLISIONS * sizeof(float));

    // This is where the acceleration structure (BVH) is actually usefull
    query(tree, ray, candidates);

    if (candidates.count == 0) {
        return 0.0;
    }

    // Test the candidates for actual intersections
    for (int i = 0; i < candidates.count; i++) {
        int primIndex = candidates.collisions[i]*3;
        
        float4 V1 = vertices[primIndex];
        float4 V2 = vertices[primIndex + 1];
        float4 V3 = vertices[primIndex + 2];

        float t;
        if (ray.intersects(V1, V2, V3, t)) {
            candidates.collisions[tvalues.count] = candidates.collisions[i];
            tvalues.collisions[tvalues.count++] = t;
        }
    }

    if (tvalues.count < 2) {
        return 0.0;
    }

    // printf ("Tvalues count: %d\n", tvalues.count);

    // arg sort the tvalues
    int index[MAX_COLLISIONS];
    thrust::stable_sort_by_key(thrust::seq, tvalues.collisions, tvalues.collisions + tvalues.count, candidates.collisions);

    // compute the thickness
    float val = matchOuterPairs (candidates, tvalues, ray, vertices, normals, tree);
    // float val = sumTvalues(tvalues);
    return val;
}

__device__ float traceRay (
        Ray &ray, Tree &tree, 
        float4* __restrict__ vertices,
        float4* __restrict__ normals) {
    CandidateList candidates;
    candidates.count = 0;
    memset(candidates.collisions, 0, MAX_COLLISIONS * sizeof(int));

    CollisionList tvalues;
    tvalues.count = 0;
    memset(tvalues.collisions, 0, MAX_COLLISIONS * sizeof(float));

    // This is where the acceleration structure (BVH) is actually usefull
    query(tree, ray, candidates);

    if (candidates.count == 0) {
        return 0.0;
    }

    // Test the candidates for actual intersections
    for (int i = 0; i < candidates.count; i++) {
        int primIndex = candidates.collisions[i]*3;
        
        float4 V1 = vertices[primIndex];
        float4 V2 = vertices[primIndex + 1];
        float4 V3 = vertices[primIndex + 2];

        float t;
        if (ray.intersects(V1, V2, V3, t)) {
            candidates.collisions[tvalues.count] = candidates.collisions[i];
            tvalues.collisions[tvalues.count++] = t;
        }
    }

    if (tvalues.count < 2) {
        return 0.0;
    }

    // printf ("Tvalues count: %d\n", tvalues.count);

    // arg sort the tvalues
    int index[MAX_COLLISIONS];
    thrust::stable_sort_by_key(thrust::seq, tvalues.collisions, tvalues.collisions + tvalues.count, candidates.collisions);

    // compute the thickness
    float val = matchOuterPairs (candidates, tvalues, ray, vertices, normals, tree);
    // float val = sumTvalues(tvalues);
    return val;
}

extern "C" __global__ void calculateBbBoxKernel (float4 *vertices, float4 *bbMin, float4 *bbMax, unsigned int nb_keys) {
    unsigned int tid = threadIdx.x + blockIdx.x * blockDim.x;

    while (tid < nb_keys) {
        float4 V1 = vertices[tid * 3];
        float4 V2 = vertices[tid * 3 + 1];
        float4 V3 = vertices[tid * 3 + 2];
        calculateTriangleBoundingBox (V1, V2, V3, bbMin[tid], bbMax[tid]);

        tid += blockDim.x * gridDim.x;
    }
    
}


extern "C" __global__ void projectTriangleCentroid(
    unsigned int const nb_keys, float4 const *vertices, unsigned int *keys,
    float4 *bbMin, float4 *bbMax, float4 const scene_bbMin, float4 const scene_bbMax) {

    unsigned int index = threadIdx.x + blockIdx.x * blockDim.x;


    while (index < nb_keys) {
        // Get the triangle vertices
        float4 V1 = vertices[index * 3];
        float4 V2 = vertices[index * 3 + 1];
        float4 V3 = vertices[index * 3 + 2];

        // Calculate the bounding box of the triangle
        calculateTriangleBoundingBox(V1, V2, V3, bbMin[index], bbMax[index]);

        // Calculate the centroid of the AABB
        float4 centroid = getBoundingBoxCentroid(bbMin[index], bbMax[index]);
        
        float4 normalizedCentroid = normalize(centroid, scene_bbMin, scene_bbMax);

        // Calculate the morton code of the triangle
        morton_t mortonCode = calculateMortonCode(normalizedCentroid);

        // Store the morton code
        keys[index] = mortonCode;

        index += blockDim.x * gridDim.x;
    }
}

extern "C" __global__ void growTreeKernel (
    unsigned int nb_keys, unsigned int *keys, unsigned int *permutation, 
    int *rope, int *left, int *entered,
    float4 *bboxMin, float4 *bboxMax) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;

    while (index < nb_keys) {
        Tree tree;
        tree.nb_keys = nb_keys;
        tree.keys = keys;
        tree.indices = permutation;
        tree.entered = entered;
        tree.rope = rope;
        tree.left = left;
        tree.bboxMin = bboxMin;
        tree.bboxMax = bboxMax;

        updateParents(tree, index);
        index += blockDim.x * gridDim.x;
    }
}

// extern "C" __global__ void growVoxelgridTreeKernel (
//     uint4 N, float4 L // voxelgrid parameters
//     unsigned int *keys, unsigned int *permutation, // tree parameters
//     int *rope, int *left, int *entered,
//     ) {
//     int index = threadIdx.x + blockIdx.x * blockDim.x;

//     while (index < nb_keys) {
//         Tree tree;
//         tree.nb_keys = N.x * N.y * N.z;
//         tree.keys = keys;
//         tree.indices = permutation;
//         tree.entered = entered;
//         tree.rope = rope;
//         tree.left = left;

//         updateParents(tree, index);
//         index += blockDim.x * gridDim.x;
//     }
// }

// extern "C" __global__ void projectNbCandidatesKernel (
//     unsigned int nb_keys,
//     float *image, uint2 N, float2 D, // image parameters
//     float4 U, float4 V, float4 W, float4 origin, // projection basis and origin
//     int *rope, int *left, unsigned *permutation,  // BVH tree
//     float4 *bboxMin, float4 *bboxMax, float4 *vertices, float4 *normals // ray casting
//     ) {
   
//     int gid_x = blockIdx.x * blockDim.x + threadIdx.x;
//     int gid_y = blockIdx.y * blockDim.y + threadIdx.y;

//     if (N.x == 0 || N.y == 0) {
//         return;
//     }

//     if (gid_x >= N.x || gid_y >= N.y) {
//         return;
//     }

//     Tree tree;
//     tree.nb_keys = nb_keys;
//     tree.rope = rope;
//     tree.left = left;
//     tree.indices = permutation;
//     tree.bboxMin = bboxMin;
//     tree.bboxMax = bboxMax;

//     for (int i = gid_x; i < N.x; i += blockDim.x * gridDim.x) {
//         for (int j = gid_y; j < N.y; j += blockDim.y * gridDim.y) {

//             float4 point_local_basis = phi(i, j, D, N);
//             float4 point_new_basis = origin + U * point_local_basis.x + V * point_local_basis.y;
//             Ray ray = Ray(point_new_basis, W);

//             float thickness = getNbCandidate (ray, tree, vertices, normals);

//             image[j * N.x + i] = thickness;
//         }
//     }
// }

// extern "C" __global__ void projectTvaluesKernel (
//     unsigned int nb_keys,
//     float *image, uint2 N, float2 D, // image parameters
//     float4 U, float4 V, float4 W, float4 origin, // projection basis and origin
//     int *rope, int *left, unsigned *permutation,  // BVH tree
//     float4 *bboxMin, float4 *bboxMax, float4 *vertices // ray casting
//     ) {
   
//     int gid_x = blockIdx.x * blockDim.x + threadIdx.x;
//     int gid_y = blockIdx.y * blockDim.y + threadIdx.y;

//     if (N.x == 0 || N.y == 0) {
//         return;
//     }

//     if (gid_x >= N.x || gid_y >= N.y) {
//         return;
//     }

//     Tree tree;
//     tree.nb_keys = nb_keys;
//     tree.rope = rope;
//     tree.left = left;
//     tree.indices = permutation;
//     tree.bboxMin = bboxMin;
//     tree.bboxMax = bboxMax;

//     for (int i = gid_x; i < N.x; i += blockDim.x * gridDim.x) {
//         for (int j = gid_y; j < N.y; j += blockDim.y * gridDim.y) {

//             float4 point_local_basis = phi(i, j, D, N);
//             float4 point_new_basis = origin + U * point_local_basis.x + V * point_local_basis.y;
//             Ray ray = Ray(point_new_basis, W);

//             float thickness = getSumedTvalues (ray, tree, vertices);

//             image[j * N.x + i] = thickness;
//         }
//     }
// }

extern "C" __global__ void projectParallelKernel (
    unsigned nb_keys, float* image, uint2 N,
    float4 U, float4 V, float4 W,  // projection basis and origin
    float4 upperleft_origin, float2 ps,
    int* rope,
    int* left,
    unsigned* permutation,  // BVH tree
    float4* bboxMin,
    float4* bboxMax,
    float4* __restrict__ vertices,
    float4* __restrict__ normals // ray casting
    ) {
   
    int gid_x = blockIdx.x * blockDim.x + threadIdx.x;
    int gid_y = blockIdx.y * blockDim.y + threadIdx.y;

    if (N.x == 0 || N.y == 0) {
        return;
    }

    if (gid_x >= N.x || gid_y >= N.y) {
        return;
    }

    Tree tree;
    tree.nb_keys = nb_keys;
    tree.rope = rope;
    tree.left = left;
    tree.indices = permutation;
    tree.bboxMin = bboxMin;
    tree.bboxMax = bboxMax;

    float4 scaled_U = U * ps.x;
    float4 scaled_V = V * ps.y;

    for (int i = gid_x; i < N.x; i += blockDim.x * gridDim.x) {
        for (int j = gid_y; j < N.y; j += blockDim.y * gridDim.y) {
            // U, V are the scaled basis vectors for the image plane
            float4 pixel_coordinates = upperleft_origin - scaled_U * i - scaled_V * j;
            Ray ray = Ray(pixel_coordinates, W);
            image[j * N.x + i] = traceParallelRay (ray, tree, vertices, normals);
        }
    }
}

// extern "C" __global__ void projectGivenRaysKernel (
//     unsigned int nb_keys, unsigned int nb_rays,
//     float *ray_retvals, // projected thicknesses
//     float4 *origins, // ray origins
//     int *rope, int *left, unsigned *permutation,  // BVH tree
//     float4 *bboxMin, float4 *bboxMax, float4 *vertices, float4 *normals // ray casting
//     ) {
   
//     int gid_x = blockIdx.x * blockDim.x + threadIdx.x;

//     if (gid_x >= nb_rays) {
//         return;
//     }

//     Tree tree;
//     tree.nb_keys = nb_keys;
//     tree.rope = rope;
//     tree.left = left;
//     tree.indices = permutation;
//     tree.bboxMin = bboxMin;
//     tree.bboxMax = bboxMax;

//     float4 W = make_float4(0.0, 0.0, -1.0, 0.0);

//     for (int i = gid_x; i < nb_rays; i += blockDim.x * gridDim.x) {        
//         Ray ray = Ray(origins[i], W);
//         ray_retvals[i] = traceParallelRay (ray, tree, vertices, normals);
//     }
// }

extern "C" __global__ void projectPerspectiveKernel (
    unsigned nb_keys, float* image, uint2 N,
    float4 U, float4 V, float4 W,  // projection basis and origin
    float4 upperleft_origin, float4 ray_origin, float2 ps, 
    int* rope,
    int* left,
    unsigned* permutation,  // BVH tree
    float4* bboxMin,
    float4* bboxMax,
    float4* __restrict__ vertices,
    float4* __restrict__ normals // ray casting
    ) {
   
    int gid_x = blockIdx.x * blockDim.x + threadIdx.x;
    int gid_y = blockIdx.y * blockDim.y + threadIdx.y;

    if (N.x == 0 || N.y == 0) {
        return;
    }

    if (gid_x >= N.x || gid_y >= N.y) {
        return;
    }

    Tree tree;
    tree.nb_keys = nb_keys;
    tree.rope = rope;
    tree.left = left;
    tree.indices = permutation;
    tree.bboxMin = bboxMin;
    tree.bboxMax = bboxMax;

    float4 scaled_U = U * ps.x;
    float4 scaled_V = V * ps.y;
    

    for (int i = gid_x; i < N.x; i += blockDim.x * gridDim.x) {
        for (int j = gid_y; j < N.y; j += blockDim.y * gridDim.y) {
            // U, V are the scaled basis vectors for the image plane
            float4 pixel_coordinates = upperleft_origin - scaled_U * i - scaled_V * j;
            float4 direction = pixel_coordinates - ray_origin;
            Ray ray = Ray(ray_origin, direction);
            // ray.print();

            float thickness = traceRay (ray, tree, vertices, normals);

            image[j * N.x + i] = thickness;
        }
    }
}